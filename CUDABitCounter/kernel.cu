#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "math_functions.h"
#include ""
#include <time.h>

extern "C"
{
	__declspec(dllexport) void BitCounter(int* arr, int x, int y, int* output);
}

__device__ int CountBits(int x)
        {
            x -= (x >> 1) & (0x55555555);
            x = (x & 0x33333333) + ((x >> 2) & 0x33333333);
            x = (x + (x >> 4)) & 0x0F0F0F0F;
            x = (x & 0x00FF00FF) + ((x >> 8) & 0x00FF00FF);
            x = (x & 0x0000FFFF) + (x >> 16);
            return x;
		}

__global__ void CountBitsInMatrix(int *arr, int x, int y, int pitch, int *res)
		{

			int sum = 0;
			
			const int length = pitch * y;
			
			int Id = threadIdx.x + blockIdx.x * blockDim.x;
			
			if (Id < x)
			{
				int accum = 0;
				int tmp = Id;
				while (tmp < length)
				{
					accum += CountBits(arr[tmp]);
					tmp += pitch;
				}
				res[Id] = accum;
			}
		}
	 
	void BitCounter(int* arr, int x, int y, int* output)
		{
			int *res, *resDev, *arrDev; 
			size_t pitch;
			
			hipError_t status;
			status = hipMalloc( (void**)&resDev, x * sizeof(int) );
			status = hipMallocPitch((void**)&arrDev, &pitch, x * sizeof(int), y);
			
			status = hipMemcpy2D(arrDev, pitch, arr, x * sizeof(int), x * sizeof(int), y, hipMemcpyHostToDevice);
			pitch /= sizeof(int);

			dim3 amountBlock = dim3((x + 1023) / 1024);
			dim3 amountTrds = dim3(1024);
			CountBitsInMatrix<<<amountBlock, amountTrds>>>(arrDev, x, y, pitch, resDev);
			status = hipDeviceSynchronize();
			res = (int*)malloc( x * sizeof(int));
			hipMemcpy(res, resDev, x * sizeof(int), hipMemcpyDeviceToHost);
			
			hipFree(resDev);
			hipFree(arrDev);
			
			output = res;
		}
	
	main()
	{
		
	}