﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "ConvolutionHelper.h"

extern "C"{

__declspec(dllexport) int main();

}

int main()
{
	// Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

	FILE* f = fopen("C:\\temp\\104_6.bin","rb");

	int width;
	int height;

	fread(&width,sizeof(int),1,f);
	fread(&height,sizeof(int),1,f);

	int* ar = (int*)malloc(sizeof(int)*width*height);
	float* ar2 = (float*)malloc(sizeof(float)*width*height);

	fread(ar,sizeof(int),width*height,f);
	for(int i=0;i<width*height;i++)
	{
		ar2[i]=ar[i];
	}
	fclose(f);
	CUDAArray<float> sourceImage = CUDAArray<float>(ar2,width,height);

	CUDAArray<float> g1 = Reduce(sourceImage,1.7f);
	CUDAArray<float> g2 = Reduce(g1,1.21f);
	CUDAArray<float> g3 = Reduce(g2,1.3f);
	CUDAArray<float> g4 = Reduce(g3,1.3f);

	CUDAArray<float> p3 = Expand(g4, 1.3f, g3.Width, g3.Height);
	CUDAArray<float> p2 = Expand(g3, 1.3f, g2.Width, g2.Height);
	CUDAArray<float> p1 = Expand(g2, 1.21f,g1.Width, g1.Height);

	SubtractArray(g3,p3);
	SubtractArray(g2,p2);
	SubtractArray(g1,p1);

	float* arTest = p1.GetData();
	f = fopen("C:\\temp\\104_6_e1.bin","wb");
	fwrite(&p1.Width,sizeof(int),1,f);
	fwrite(&p1.Height,sizeof(int),1,f);
	for(int i=0;i<p1.Width*p1.Height;i++)
	{
		int value = (int)arTest[i];
		int result = fwrite(&value,sizeof(int),1,f);
		result++;
	}
	fclose(f);
	sourceImage.Dispose();
	g1.Dispose();
	g2.Dispose();
	g3.Dispose();
	g4.Dispose();
	p1.Dispose();
	p2.Dispose();
	p3.Dispose();
	free(ar);
	free(ar2);
	free(arTest);
	hipDeviceReset();
    return 0;
}