#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "MinutiaExtraction.h"
#include <time.h>
extern "C"{

__declspec(dllexport) int main();

}

const float tau1 = 0.1f;
const float tau2 = 0.3f;

CUDAArray<float> EnhanceImage(CUDAArray<float> sourceImage)
{
	CUDAArray<float> g1 = Reduce(sourceImage,1.7f);
	CUDAArray<float> g2 = Reduce(g1,1.21f);
	CUDAArray<float> g3 = Reduce(g2,1.3f);
	CUDAArray<float> g4 = Reduce(g3,1.3f);

	CUDAArray<float> p3 = Expand(g4, 1.3f, g3.Width, g3.Height);
	CUDAArray<float> p2 = Expand(g3, 1.3f, g2.Width, g2.Height);
	CUDAArray<float> p1 = Expand(g2, 1.21f,g1.Width, g1.Height);
	
	SubtractArray(g3,p3);
	EnhanceContrast(g3);
	SubtractArray(g2,p2);
	EnhanceContrast(g2);
	SubtractArray(g1,p1);
	EnhanceContrast(g1);

	CUDAArray<float> ls1Real;
	CUDAArray<float> ls1Im;
	EstimateLS(&ls1Real, &ls1Im, g1, 0.6f, 3.2f);
	
	CUDAArray<float> ls2Real;
	CUDAArray<float> ls2Im;
	EstimateLS(&ls2Real, &ls2Im, g2, 0.6f, 3.2f);

	CUDAArray<float> ls3Real;
	CUDAArray<float> ls3Im;
	EstimateLS(&ls3Real, &ls3Im, g3, 0.6f, 3.2f);

	CorrectLS1WithLS2(ls1Real, ls1Im, ls2Real, ls2Im);
	
	DirectionFiltering(g1, ls1Real, ls1Im, tau1, tau2);
	
	DirectionFiltering(g2, ls2Real, ls2Im, tau1, tau2);
	
	DirectionFiltering(g3, ls3Real, ls3Im, tau1, tau2);

	CUDAArray<float> el3 = Expand(g3, 1.3f, g2.Width, g2.Height);
	AddArray(el3,g2);
	CUDAArray<float> el2 = Expand(el3, 1.21f,g1.Width, g1.Height);
	el3.Dispose();
	AddArray(el2,g1);
	CUDAArray<float> enhanced = Expand(el2, 1.7f,sourceImage.Width, sourceImage.Height);
	el2.Dispose();
	EnhanceContrast(enhanced);

	FixValues(enhanced);

	g1.Dispose();
	g2.Dispose();
	g3.Dispose();
	g4.Dispose();
	p1.Dispose();
	p2.Dispose();
	p3.Dispose();
	return enhanced;
}

CUDAArray<float> loadImage(const char* name, bool sourceIsFloat = false)
{
	FILE* f = fopen(name,"rb");
			
	int width;
	int height;
	
	fread(&width,sizeof(int),1,f);
			
	fread(&height,sizeof(int),1,f);
	
	float* ar2 = (float*)malloc(sizeof(float)*width*height);

	if(!sourceIsFloat)
	{
		int* ar = (int*)malloc(sizeof(int)*width*height);
		fread(ar,sizeof(int),width*height,f);
		for(int i=0;i<width*height;i++)
		{
			ar2[i]=ar[i];
		}
		
		free(ar);
	}
	else
	{
		fread(ar2,sizeof(float),width*height,f);
	}
	
	fclose(f);

	CUDAArray<float> sourceImage = CUDAArray<float>(ar2,width,height);

	free(ar2);		

	return sourceImage;
}

int main()
{
	// Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

	char* buf = (char*)malloc(sizeof(char)*50);
	float time = 0;
	//for(int  i=1;i<=110;i++)
	//{
	//	for(int j=1;j<=8;j++)
	//	{
	//		sprintf(buf,"C:\\temp\\bin\\%d_%d.bin",i,j);

	//		hipDeviceReset();
	//		FillDirections();

	//		CUDAArray<float> sourceImage = loadImage(buf);

	//		clock_t t1 = clock();

	//		CUDAArray<float> enhanced = EnhanceImage(sourceImage);

	//		clock_t t2 = clock();
	//		
	//		float dt = (float)(t2-t1) / CLOCKS_PER_SEC;
	//		time+=dt;
	//		sprintf(buf,"C:\\temp\\enh\\%d_%d.bin",i,j);

	//		SaveArray(enhanced, buf);
	//		sourceImage.Dispose();
	//		enhanced.Dispose();
	//	}
	//}

	//time /= 880;

	//// minutia extraction

	//time = 0;

	//for(int  i=1;i<=110;i++)
	//{
	//	for(int j=1;j<=8;j++)
	//	{
	//		hipDeviceReset();

	//		sprintf(buf,"C:\\temp\\enh\\%d_%d.bin",i,j);

	//		CUDAArray<float> sourceImage = loadImage(buf,true);
	//			
	//		clock_t t1 = clock();

	//		int* xs;
	//		int* ys;

	//		ExtractMinutiae(&xs, &ys, sourceImage);

	//		time+= (float)(clock()-t1) / CLOCKS_PER_SEC;

	//		sprintf(buf,"C:\\temp\\min\\%d_%d.min",i,j);
	//		FILE* f = fopen(buf,"wb");
	//		int y = 32;
	//		fwrite(&y,sizeof(int),1,f);
	//		for(int i=0;i<32;i++)
	//		{
	//			int result = fwrite(xs+i,sizeof(int),1,f);
	//			result = fwrite(ys+i,sizeof(int),1,f);
	//		}
	//		fclose(f);
	//		free(xs);
	//		free(ys);

	//		sourceImage.Dispose();
	//	}
	//}

	//time /= 880;

	//// minutia matching
	
	int* dBaseX = (int*)malloc(sizeof(int)*32*880);
	int* dBaseY = (int*)malloc(sizeof(int)*32*880);
	
	int ptrX = 0, ptrY = 0;

	for(int  i=1;i<=110;i++)
	{
		for(int j=1;j<=8;j++)
		{
			sprintf(buf,"C:\\temp\\min\\%d_%d.min",i,j);

			FILE* f = fopen(buf,"rb");

			int amount = 0;

			fread(&amount, sizeof(int), 1, f);
			
			for(int n = 0; n< amount; n++)
			{
				fread(dBaseX+ptrX++,sizeof(int), 1,f);
				fread(dBaseY+ptrY++,sizeof(int), 1,f);
			}

			fclose(f);
		}
	}

	int* same = (int*)malloc(sizeof(int)*33);
	memset(same,0,33*sizeof(int));

	int* different = (int*)malloc(sizeof(int)*33);
	memset(different,0,33*sizeof(int));

	time = 0;

	int totalMatches = 0;

	for(int i=0; i<879; i++)
	{
		hipDeviceReset();

		int matches = 879-i;

		if(matches%100==0)printf("DBASE size is %d\n",matches);

		totalMatches+=matches;

		CUDAArray<int> cudaBaseX = CUDAArray<int>(dBaseX+32*(1+i),32,879-i);
		CUDAArray<int> cudaBaseY = CUDAArray<int>(dBaseY+32*(1+i),32,879-i);

		clock_t t1 = clock();

		CUDAArray<int> result = MatchFingers(dBaseX+32*i,dBaseY+32*i, cudaBaseX, cudaBaseY);

		int* resultLocal = result.GetData();

		time+= clock()-t1;

		for(int j=0;j<result.Width*result.Height;j++)
		{
			if(i/8 == (i+1+j)/8)same[resultLocal[j]]++;
			else different[resultLocal[j]]++;
		}

		cudaBaseX.Dispose();
		cudaBaseY.Dispose();
		result.Dispose();
		free(resultLocal);
	}

	time /= CLOCKS_PER_SEC;
	time /= totalMatches;

	free(dBaseX);
	free(dBaseY);

	FILE* f1 = fopen("C:\\temp\\ZeeBigResult.bin","wb");
	
	fwrite(same, sizeof(int), 33, f1);
	fwrite(different, sizeof(int), 33, f1);

	fclose(f1);

	hipDeviceReset();
	free(buf);
    return 0;
}