
#include "hip/hip_runtime.h"


#include <stdio.h>
#include<stdlib.h>
//#include<MinutiaMatching.h>

//cudaError_t addWithCuda(int *c, const int *a, const int *b, size_t size);
hipError_t addWithCuda(double **picture, int size, double **result);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}

//CUDAArray<float> loadImage(const char* name, bool sourceIsFloat = false)
//{
//	FILE* f = fopen(name,"rb");
//			
//	int width;
//	int height;
//	
//	fread(&width,sizeof(int),1,f);
//			
//	fread(&height,sizeof(int),1,f);
//	
//	float* ar2 = (float*)malloc(sizeof(float)*width*height);
//
//	if(!sourceIsFloat)
//	{
//		int* ar = (int*)malloc(sizeof(int)*width*height);
//		fread(ar,sizeof(int),width*height,f);
//		for(int i=0;i<width*height;i++)
//		{
//			ar2[i]=ar[i];
//		}
//		
//		free(ar);
//	}
//	else
//	{
//		fread(ar2,sizeof(float),width*height,f);
//	}
//	
//	fclose(f);
//
//	CUDAArray<float> sourceImage = CUDAArray<float>(ar2,width,height);
//
//	free(ar2);		
//
//	return sourceImage;
//}


__device__ double B(double *picture, int x, int y, size_t pitch)        //����� �(�) ���������� ���������� ������ �������� � ����������� ����� �
{
	return picture[x + (y - 1)*pitch] + picture[x + 1 + (y - 1)*pitch] + picture[x + 1 + y*pitch] + picture[x + 1 + (y + 1)*pitch] +
           picture[x * (y + 1)*pitch] + picture[x - 1 + (y + 1)*pitch] + picture[x - 1 + y*pitch] + picture[x - 1 * (y - 1)*pitch];
}

__device__ double A(double *picture, int x, int y, size_t pitch)        //����� �(�) ���������� ���������� ������ ������ ����� � ������ �������� ������ ����� � (..0->1..)
{
	int counter = 0;
    if((picture[x + (y - 1)*pitch] == 0) && (picture[x + 1 + (y - 1)*pitch] == 1))
    {
        counter++;
    }
    if ((picture[x + 1 + (y - 1)*pitch] == 0) && (picture[x + 1 + y*pitch] == 1))
    {
        counter++;
    }
    if ((picture[x + 1 + y*pitch] == 0) && (picture[x + 1 + (y + 1)*pitch] == 1))
    {
        counter++;
    }
    if ((picture[x + 1 + (y + 1)*pitch] == 0) && (picture[x + (y + 1)*pitch] == 1))
    {
        counter++;
    }
    if ((picture[x + (y + 1)*pitch] == 0) && (picture[x - 1 + (y + 1)*pitch] == 1))
    {
        counter++;
    }
    if ((picture[x - 1 + (y + 1)*pitch] == 0) && (picture[x - 1 + y*pitch] == 1))
    {
        counter++;
    }
    if ((picture[x - 1 + y*pitch] == 0) && (picture[x - 1 + (y - 1)*pitch] == 1))
    {
        counter++;
    }
    if ((picture[x - 1 + (y - 1)*pitch] == 0) && (picture[x + (y - 1)*pitch] == 1))
    {
        counter++;
    }
    return counter;
}


__global__ void ThiningPictureWithCUDA(double* newPicture,double *thinnedPicture ,size_t pitch, int width, int height)
{
	double *picture = newPicture;
	int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    if((x > 0) && (y > 0) && (x < width) && (y < height))
	{             
		if ((picture[x, y] == 1) && (2 <= B(picture, x, y, pitch)) && (B(picture, x, y, pitch) <= 6) && (A(picture, x, y, pitch) == 1) &&     //���������������� �������� �����, ��. Zhang-Suen thinning algorithm, http://www-prima.inrialpes.fr/perso/Tran/Draft/gateway.cfm.pdf
            (picture[x + (y - 1)*pitch]*picture[x + 1 + y*pitch]*picture[x + (y + 1)*pitch] == 0) &&
            (picture[x + 1 + y*pitch]*picture[x + (y + 1)*pitch]*picture[x - 1 + y*pitch] == 0))
        {
            picture[x + y*pitch] = 0;
        }
		
		if ((picture[x + y*pitch] == 1) && (2 <= B(picture, x, y, pitch)) && (B(picture, x, y, pitch) <= 6) && (A(picture, x, y, pitch) == 1) &&
			(picture[x + (y - 1)*pitch] * picture[x + 1 + y*pitch] * picture[x - 1 + y*pitch] == 0) &&
			(picture[x * (y - 1)*pitch] * picture[x + (y + 1)*pitch] * picture[x - 1 + y*pitch] == 0))
		{
			picture[x + y*pitch] = 0;
		} 
		
		if ((picture[x, y] == 1) &&
            (((picture[x, (y - 1)*pitch] * picture[x + 1 + y*pitch] == 1) && (picture[x - 1 + (y + 1)*pitch] != 1)) || ((picture[x + 1 + y*pitch] * picture[x + (y + 1)*pitch] == 1) && (picture[x - 1 + (y - 1)*pitch] != 1)) ||      //��������� ����������� ��������� ��� ��� �������� ����������
            ((picture[x + (y + 1)*pitch] * picture[x - 1 + y*pitch] == 1) && (picture[x + 1 + (y - 1)*pitch] != 1)) || ((picture[x + (y - 1)*pitch] * picture[x - 1 + y*pitch] == 1) && (picture[x + 1 + (y + 1)*pitch] != 1))))
        {
            picture[x + y*pitch] = 0;
        }
		
		thinnedPicture = picture;
	}
}








int main()
{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };

    // Add vectors in parallel.
	int size = 32;
	double **picture = (double**)malloc(size*size*sizeof(double*));
	for(int i = 0; i < size; i++){
		picture[i] = (double*)malloc(size*sizeof(double));
	}
	double **result = (double**)malloc(size*size*sizeof(double*));
	for(int i = 0; i < size; i++){
		result[i] = (double*)malloc(size*sizeof(double));
	}
	for(int i = 0; i < size; i++)
	{
		for(int j = 0; j < size; j++)
		{
			scanf("%d",&picture[i][j]);
		}
	}

    hipError_t cudaStatus = addWithCuda(picture, size, result);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = cudaDeviceReset();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceReset failed!");
//        return 1;
//    }
	for(int i = 0; i < size; i++){
		free(picture[i]);
	}
	free(picture);\
	for(int i = 0; i < size; i++){
		free(result[i]);
	}
	free(result);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double **picture, int size, double **result)
{
    //int *dev_a = 0;
    //int *dev_b = 0;
    //int *dev_c = 0;
	double* dev_picture;
	double* dev_pictureThinned;
	int width, height;
	width = size;
	height = size;

    hipError_t cudaStatus;
	size_t pitch;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	//Allocate GPU buffers for picture.
	hipError_t cudastatus;
	cudaStatus = hipMallocPitch((void**)&dev_picture, &pitch, width*sizeof(int), height);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMallocPitch!");
        goto Error;
    }
    //cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMalloc failed!");
    //    goto Error;
    //}

    //cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMalloc failed!");
    //    goto Error;
    //}

    // Copy input vpicture from host memory to GPU buffers.
    cudaStatus = hipMemcpy2D(dev_picture, pitch, picture, width*sizeof(int), width*sizeof(int), height, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMemcpy failed!");
    //    goto Error;
    //}

    //cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMemcpy failed!");
    //    goto Error;
    //}

    // Launch a kernel on the GPU with one thread for each element.
    ThiningPictureWithCUDA<<<1, size>>>(dev_picture, dev_pictureThinned, pitch, width, height);

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	cudastatus = hipMemcpy2D(result,width*sizeof(int),dev_pictureThinned,pitch,width*sizeof(int),height,hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_picture);
    hipFree(dev_pictureThinned);
    //cudaFree(dev_b);
    
    return cudaStatus;
}
