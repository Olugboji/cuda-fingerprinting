#include "hip/hip_runtime.h"
﻿extern "C"{

__declspec(dllexport) void Init();

__declspec(dllexport) void FillDirections();

__declspec(dllexport) int Identify(float* image, int width, int height);

__declspec(dllexport) void Enhance(float* image, int width, int height);
}

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "MinutiaExtraction.h"
#include <time.h>

const float tau1 = 0.1f;
const float tau2 = 0.3f;

CUDAArray<float> EnhanceImage(CUDAArray<float> sourceImage)
{
	CUDAArray<float> g1 = Reduce(sourceImage,1.7f);
	CUDAArray<float> g2 = Reduce(g1,1.21f);
	CUDAArray<float> g3 = Reduce(g2,1.3f);
	CUDAArray<float> g4 = Reduce(g3,1.3f);

	CUDAArray<float> p3 = Expand(g4, 1.3f, g3.Width, g3.Height);
	CUDAArray<float> p2 = Expand(g3, 1.3f, g2.Width, g2.Height);
	CUDAArray<float> p1 = Expand(g2, 1.21f,g1.Width, g1.Height);
	
	SubtractArray(g3,p3);
	EnhanceContrast(g3);
	SubtractArray(g2,p2);
	EnhanceContrast(g2);
	SubtractArray(g1,p1);
	EnhanceContrast(g1);

	CUDAArray<float> ls1Real;
	CUDAArray<float> ls1Im;
	EstimateLS(&ls1Real, &ls1Im, g1, 0.6f, 3.2f);
	
	CUDAArray<float> ls2Real;
	CUDAArray<float> ls2Im;
	EstimateLS(&ls2Real, &ls2Im, g2, 0.6f, 3.2f);

	CUDAArray<float> ls3Real;
	CUDAArray<float> ls3Im;
	EstimateLS(&ls3Real, &ls3Im, g3, 0.6f, 3.2f);

	CorrectLS1WithLS2(ls1Real, ls1Im, ls2Real, ls2Im);
	
	DirectionFiltering(g1, ls1Real, ls1Im, tau1, tau2);
	
	DirectionFiltering(g2, ls2Real, ls2Im, tau1, tau2);
	
	DirectionFiltering(g3, ls3Real, ls3Im, tau1, tau2);

	CUDAArray<float> el3 = Expand(g3, 1.3f, g2.Width, g2.Height);
	AddArray(el3,g2);
	CUDAArray<float> el2 = Expand(el3, 1.21f,g1.Width, g1.Height);
	el3.Dispose();
	AddArray(el2,g1);
	CUDAArray<float> enhanced = Expand(el2, 1.7f,sourceImage.Width, sourceImage.Height);
	el2.Dispose();
	EnhanceContrast(enhanced);

	FixValues(enhanced);

	g1.Dispose();
	g2.Dispose();
	g3.Dispose();
	g4.Dispose();
	p1.Dispose();
	p2.Dispose();
	p3.Dispose();
	return enhanced;
}

void Enhance(float* image, int width, int height)
{
	hipSetDevice(0);
	FillDirections();
	CUDAArray<float> arr = CUDAArray<float>(image, width, height);
	CUDAArray<float> arr2 = EnhanceImage(arr);
	arr2.GetData(image);
	arr.Dispose();
	arr2.Dispose();
	//hipDeviceReset();
}

CUDAArray<float> loadImage(const char* name, bool sourceIsFloat = false)
{
	FILE* f = fopen(name,"rb");
			
	int width;
	int height;
	
	fread(&width,sizeof(int),1,f);
			
	fread(&height,sizeof(int),1,f);
	
	float* ar2 = (float*)malloc(sizeof(float)*width*height);

	if(!sourceIsFloat)
	{
		int* ar = (int*)malloc(sizeof(int)*width*height);
		fread(ar,sizeof(int),width*height,f);
		for(int i=0;i<width*height;i++)
		{
			ar2[i]=ar[i];
		}
		
		free(ar);
	}
	else
	{
		fread(ar2,sizeof(float),width*height,f);
	}
	
	fclose(f);

	CUDAArray<float> sourceImage = CUDAArray<float>(ar2,width,height);

	free(ar2);		

	return sourceImage;
}

int main()
{
	Init();

	CUDAArray<float> img = loadImage("C:\\temp\\bin\\104_6.bin");

	float* image = img.GetData();

	for(int i=0;i<img.Height*img.Width;i++)
	{
		if(image[i]<0||image[i]>255)
		{
			i++;
		}
	}
	clock_t clk = clock();
	FillDirections();
	Enhance(image, img.Width, img.Height);
	clk = clock() - clk;
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipSetDevice(0);

	char* buf = (char*)malloc(sizeof(char)*50);
	float time = 0;
	//for(int  i=1;i<=110;i++)
	//{
	//	for(int j=1;j<=8;j++)
	//	{
	//		sprintf(buf,"C:\\temp\\bin\\%d_%d.bin",i,j);

	//		hipDeviceReset();
	//		FillDirections();

	//		CUDAArray<float> sourceImage = loadImage(buf);

	//		clock_t t1 = clock();

	//		CUDAArray<float> enhanced = EnhanceImage(sourceImage);

	//		clock_t t2 = clock();
	//		
	//		float dt = (float)(t2-t1) / CLOCKS_PER_SEC;
	//		time+=dt;
	//		sprintf(buf,"C:\\temp\\enh_bin\\%d_%d.bin",i,j);

	//		SaveArray(enhanced, buf);
	//		sourceImage.Dispose();
	//		enhanced.Dispose();
	//	}
	//}

	//time /= 880;

	// minutia extraction

	time = 0;

	for(int  i=1;i<=110;i++)
	{
		for(int j=1;j<=8;j++)
		{
			hipDeviceReset();

			sprintf(buf,"C:\\temp\\enh_bin\\%d_%d.bin",i,j);

			CUDAArray<float> sourceImage = loadImage(buf,true);
				
			clock_t t1 = clock();

			int* xs;
			int* ys;

			ExtractMinutiae(&xs, &ys, sourceImage);

			time+= (float)(clock()-t1) / CLOCKS_PER_SEC;

			sprintf(buf,"C:\\temp\\min\\%d_%d.min",i,j);
			FILE* f = fopen(buf,"wb");
			int y = 32;
			fwrite(&y,sizeof(int),1,f);
			for(int i=0;i<32;i++)
			{
				int result = fwrite(xs+i,sizeof(int),1,f);
				result = fwrite(ys+i,sizeof(int),1,f);
			}
			fclose(f);
			free(xs);
			free(ys);

			sourceImage.Dispose();
		}
	}

	time /= 880;

	//// minutia matching
		int* dBaseX = (int*)malloc(sizeof(int)*32*880);
	int* dBaseY = (int*)malloc(sizeof(int)*32*880);
	
	int ptrX = 0, ptrY = 0;

	for(int  i=1;i<=110;i++)
	{
		for(int j=1;j<=8;j++)
		{
			sprintf(buf,"C:\\temp\\min\\%d_%d.min",i,j);

			FILE* f = fopen(buf,"rb");

			int amount = 0;

			fread(&amount, sizeof(int), 1, f);
			
			for(int n = 0; n< amount; n++)
			{
				fread(dBaseX+ptrX++,sizeof(int), 1,f);
				fread(dBaseY+ptrY++,sizeof(int), 1,f);
			}

			fclose(f);
		}
	}

	int* same = (int*)malloc(sizeof(int)*33);
	memset(same,0,33*sizeof(int));

	int* different = (int*)malloc(sizeof(int)*33);
	memset(different,0,33*sizeof(int));

	time = 0;

	int totalMatches = 0;

	for(int i=0; i<879; i++)
	{
		hipDeviceReset();

		int matches = 879-i;

		if(matches%100==0)printf("DBASE size is %d\n",matches);

		totalMatches+=matches;

		CUDAArray<int> cudaBaseX = CUDAArray<int>(dBaseX+32*(1+i),32,879-i);
		CUDAArray<int> cudaBaseY = CUDAArray<int>(dBaseY+32*(1+i),32,879-i);

		clock_t t1 = clock();

		CUDAArray<int> result = MatchFingers(dBaseX+32*i,dBaseY+32*i, cudaBaseX, cudaBaseY);

		int* resultLocal = result.GetData();

		time+= clock()-t1;

		for(int j=0;j<result.Width*result.Height;j++)
		{
			if(i/8 == (i+1+j)/8)same[resultLocal[j]]++;
			else different[resultLocal[j]]++;
		}

		cudaBaseX.Dispose();
		cudaBaseY.Dispose();
		result.Dispose();
		free(resultLocal);
	}

	time /= CLOCKS_PER_SEC;
	time /= totalMatches;

	free(dBaseX);
	free(dBaseY);

	FILE* f1 = fopen("C:\\temp\\ZeeBigResult.bin","wb");
	
	fwrite(same, sizeof(int), 33, f1);
	fwrite(different, sizeof(int), 33, f1);

	fclose(f1);

	hipDeviceReset();
	free(buf);
    return 0;
}



int* dBaseX;
int* dBaseY;

void Init()
{
	hipSetDevice(0);
	char* buf = (char*)malloc(sizeof(char)*50);

	dBaseX = (int*)malloc(sizeof(int)*32*440);
	dBaseY = (int*)malloc(sizeof(int)*32*440);
	
	int ptrX = 0, ptrY = 0;

	for(int  i=1;i<=110;i++)
	{
		for(int j=1;j<=4;j++)
		{
			sprintf(buf,"C:\\temp\\min\\%d_%d.min",i,j);

			FILE* f = fopen(buf,"rb");

			int amount = 0;

			fread(&amount, sizeof(int), 1, f);
			
			for(int n = 0; n< amount; n++)
			{
				fread(dBaseX+ptrX++,sizeof(int), 1,f);
				fread(dBaseY+ptrY++,sizeof(int), 1,f);
			}

			fclose(f);
		}
	}
	free(buf);
}

int Identify(float* image, int width, int height)
{
	hipDeviceReset();

	CUDAArray<float> cudaImg = CUDAArray<float>(image, width, height);
	//SaveArray(cudaImg,"C:\\temp\\check.bin");
	FillDirections();
	CUDAArray<float> enh = EnhanceImage(cudaImg);
	//SaveArray(enh,"C:\\temp\\check.bin");
	int* xs;
	int* ys;

	ExtractMinutiae(&xs, &ys, enh);

	CUDAArray<int> cudaBaseX = CUDAArray<int>(dBaseX,32,440);
	CUDAArray<int> cudaBaseY = CUDAArray<int>(dBaseY,32,440);

	CUDAArray<int> result = MatchFingers(xs,ys, cudaBaseX, cudaBaseY);

	

	int* resultLocal = result.GetData();

	int max =0;
	int index = 0;

	for(int i=0;i<440;i++)
	{
		if(resultLocal[i]>max)
		{
			max = resultLocal[i];
			index = i/4;
		}
	}

	cudaImg.Dispose();
	enh.Dispose();
	cudaBaseX.Dispose();
	cudaBaseY.Dispose();
	result.Dispose();
	free(resultLocal);
	free(xs);
	free(ys);

	return index+1;
}