#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "CUDAArray.h";
#include "ConvolutionHelper.h"

#define _USE_MATH_DEFINES
#include <math.h>

#define ceilMod(x, y) (x+y-1)/y

const int R = 70;
const int Ns = 16; // 8
const int Nd = 6;
const double SigmaS = 28 / 3;
const double SigmaD = 2* M_PI / 9;
double deltaS;
double deltaD;
const double MuPsi = 0.001;
const int BigSigma = 50;
const double MinVC = 0.75;
const double MinM = 2;
const double MinME = 0.6;
const double SigmaTetta = M_PI / 2;  
const double N = 10;
const double DictionaryCount = 360; // 720, 1440

struct Minutiae
{
	int x;
	int y;
	int numMinutiaeAround;
	float angle;
};

__global__ void cudaMCC (Minutiae* minutiae, CUDAArray<double> integralValues)
{

}

__global__ void cudaMakeTableOfIntegrals(double* integralParameters, CUDAArray<double> integralValues, 
	bool* workingArea, double factor, double h)
{
	int column = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	double a = integralParameters[column] - deltaD / 2;
	double integrand = 0;
    double result = 0;

    for (int i = 0; i < N; i++)
    {
		integrand = a + ((2 * i + 1) * h) / 2;
		integrand = exp((-integrand * integrand) / (2 * SigmaD * SigmaD));
		result += h * integrand;    
	}
	
	integralValues.SetAt(1, column, result * factor);
}

void MCCMethod(Minutiae *minutiae, int minutiaeCount, int rows, int columns)
{
	hipError_t cudaStatus = hipSetDevice(0);

	deltaS = 2 * R / Ns;
	deltaD = 2 * M_PI / Nd;
	double* integralParameters = (double*)malloc(DictionaryCount*sizeof(double));

	//------------new method--------------------
	 double key = -M_PI;
	 double step = 2 * M_PI / DictionaryCount;

	 for (int i = 0; i < DictionaryCount; i++)
	 {
		 integralParameters[i] = key;
		 key += step;
	 }
	 //----------------------------------------
	 
	 CUDAArray<double> integralValues = CUDAArray<double>(DictionaryCount, 1);
	 cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("integralValues = CUDAArray<double>(DictionaryCount, 1); - ERROR!!!\n");
	  }

	 dim3 blockSize = dim3(defaultThreadCount,defaultThreadCount);
	 dim3 gridSize = dim3(ceilMod(DictionaryCount, defaultThreadCount));
	 CUDAArray<double> integralValues = CUDAArray<double>(DictionaryCount, 1);

	 cudaMakeTableOfIntegrals<<<gridSize,blockSize>>>(integralParameters, integralValues);
	 cudaStatus = hipGetLastError();
	 if (cudaStatus != hipSuccess) 
	 {
		printf("cudaMakeTableOfIntegrals - ERROR!!!\n");
	 }

	double factor = 1 / (SigmaD * sqrt(2 * M_PI));
    double h = deltaD / N;
		 
	gridSize = dim3(ceilMod(minutiaeCount, defaultThreadCount));
	cudaMCC<<<gridSize,blockSize>>>(minutiae, integralValues, factor, h);
}

void main()
{
	// MinutiaDetectionSpecial.kernel.cu = > Minutiae *minutiae (array of Minutiae struct), int minutiaeCount(length of array)
	// CUDAConvexHull.BuildWorkingArea(int *field,int rows,int columns,int radius,int *IntMinutiae,int NoM);
	// workingArea = WorkingArea.BuildWorkingArea(minutiae, Constants.R, rows, columns);
}
