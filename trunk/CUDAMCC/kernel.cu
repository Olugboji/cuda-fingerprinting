#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "CUDAArray.h";
#include "ConvolutionHelper.h"

#define _USE_MATH_DEFINES
#include <math.h>

#define ceilMod(x, y) (x+y-1)/y

const int R = 70;
const int Ns = 16; // 8
const int Nd = 6;
const double SigmaS = 28 / 3;
const double SigmaD = 2* M_PI / 9;
const double MuPsi = 0.001;
const int BigSigma = 50;
const double MinVC = 0.75;
const double MinM = 2;
const double MinME = 0.6;
const double SigmaTetta = M_PI / 2;  
const double N = 10;
const double DictionaryCount = 360; // 720, 1440

struct Minutiae
{
	int x;
	int y;
	int numMinutiaeAround;
	float angle;
};

void MCCMethod(Minutiae *minutiae, int minutiaeCount, int rows, int columns)
{
	/*hipError_t cudaStatus = hipSetDevice(0);

	double deltaS = 2 * R / Ns;
	double deltaD = 2 * M_PI / Nd;
	double* integralParameters = (double*)malloc(DictionaryCount*sizeof(double));

	for (int i = 0; i < DictionaryCount; i++)
	{
		integralParameters[i] = 
	}

	InitialIntegralValues();

	CUDAArray<double> integralValues = CUDAArray<double>(DictionaryCount, 1);
	dim3 blockSize = dim3(defaultThreadCount,defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(DictionaryCount, defaultThreadCount));

	MakeTableOfIntegrals()*/

}

void main()
{
	// MinutiaDetectionSpecial.kernel.cu = > Minutiae *minutiae (array of Minutiae struct), int minutiaeCount(length of array)
	// CUDAConvexHull.BuildWorkingArea(int *field,int rows,int columns,int radius,int *IntMinutiae,int NoM);
}
