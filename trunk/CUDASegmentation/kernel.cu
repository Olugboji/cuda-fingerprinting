#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "ConvolutionHelper.h"
#include <stdio.h>

#define ceilMod(x, y) (x+y-1)/y

typedef struct 
	{
		int X;
		int Y;
	} Point;

typedef struct
	{
		int AreaNumber;
		int AreaSize;
		Point* Points;
	} AreaStruct;
  
/*
bool IsNearBorder(Point* points, int size, int xBorder, int yBorder)
{
	for (int i = 0; i < size; i++)
    {
		if (points[i].X == 0 || 
			points[i].Y == 0 ||
			points[i].X == xBorder || 
			points[i].Y == yBorder)
		{
			return true;
		}
	}

	return false;
}

void AddPointToArea(CUDAArray<AreaStruct> areas, int areaNumber, int pointNumber, Point newPoint)
{
	AreaStruct area = areas.At(areaNumber, 1);

	area.Points[area.AreaSize] = newPoint;
	area.AreaSize++;
	areas.SetAt(areaNumber, 1, area);
}

void MergeAreas(CUDAArray<AreaStruct> areas, AreaStruct initialValue, int maskX, int areasSize, int i, int j, int areaIndex)
{
	int areaNumberi = 0;
	int areaNumberj = 0;
	AreaStruct area = areas.At(j*maskX + i, 1);

	for (int k = 0; k < areasSize; k++)
	{
		if (area.Points[k].X == i && area.Points[k].Y == j-1)
		{
			areaNumberj = k;
		}
		if (area.Points[k].X == i-1 && area.Points[k].Y == j)
		{
			areaNumberi = k;
		}
	}
                       
	if (areaNumberi != areaNumberj)
	{
		for (int k = 0; k < (areas.At(areaNumberj, 1)).AreaSize; k++)
		{
			AddPointToArea(areas, areaNumberi, 
				(areas.At(areaNumberi, 1)).AreaSize, 
				(areas.At(areaNumberj, 1)).Points[k]
			);
		}

		areas.SetAt(areaNumberj, 1, initialValue);

		for (int k = areaNumberj + 1; k < areaIndex; k++)
		{
			areas.SetAt(k-1, 1, areas.At(k, 1));
		}
	}
    
	Point p = {i,j};
	AddPointToArea(areas, areaNumberi, (areas.At(areaNumberi, 1)).AreaSize, p);
}

bool IsLeftImageTopBlack(int i, int j, bool topValue, bool leftValue, bool isBlack) 
{
	return (j - 1 >= 0 && (topValue || isBlack) && !(topValue && isBlack) &&					//top block is black 
           (i - 1 >= 0 && (leftValue || !isBlack) && !(leftValue && !isBlack)) || i - 1 < 0);	//left block is not black or not exist
}

bool IsLeftBlackTopImage(int i, int j, bool topValue, bool leftValue, bool isBlack) 
{
	return (i - 1 >= 0 && (leftValue || isBlack) && !(leftValue && isBlack) &&					//left block is black
           (j - 1 >= 0 && (topValue || !isBlack) && !(topValue && !isBlack)) || j - 1 < 0);	    //top block is not black or not exist
}

bool IsLeftBlackTopBlack(int i, int j, bool topValue, bool leftValue, bool isBlack)
{
	return (j - 1 >= 0 && (topValue || isBlack) && !(topValue && isBlack) &&					//top block is black 
            i - 1 >= 0 && (leftValue || isBlack) && !(leftValue && isBlack));					//left block is black
}

AreaStruct MakeInitialValue(CUDAArray<AreaStruct> areas, CUDAArray<Point> initialPoints, int areasSize)
{
	Point point = {-1,-1};
	hipError_t cudaStatus;

	for (int i = 0; i < areasSize; i++)
	{
		initialPoints.SetAt(i, 1, point);
	}

	AreaStruct initialValue = {-1, 0, initialPoints.GetData()};
	
	for (int i = 0; i < areasSize; i++)
	{
		areas.SetAt(i, 1, initialValue);
	}

	return initialValue;
}

__global__ void fillArea(CUDAArray<AreaStruct> areas, int areasSize, int maskX, int iSearch, int jSearch, int i, int j, bool isFirst)
{
    int columnX = defaultColumn();
	int rowY = defaultRow();
	AreaStruct area = areas.At(rowY*maskX + columnX, 1);
	AreaStruct toSetArea;

	if (isFirst)
	{
		iSearch = i-1;
		jSearch = j;
	}
	
	for (int i = 0; i < areasSize; i++)
	{
		if (area.Points[i].X == iSearch && area.Points[i].Y == jSearch)
		{
			area.Points[area.AreaSize + 1].X = i;
			area.Points[area.AreaSize + 1].Y = j;
			area.AreaNumber++;
			toSetArea = areas.At(columnX*maskX + rowY, 1);
			toSetArea.Points = area.Points;
			areas.SetAt(columnX*maskX + rowY, 1, toSetArea);
			return;
		}
	}
}

CUDAArray<AreaStruct> GenerateAreas(CUDAArray<bool> mask, int maskX, int maskY, bool isBlack)
{
	int areasSize = maskX * maskY + 1;
	int areaIndex = 0;
	bool isLeftImageTopBlack, isLeftBlackTopImage, isLeftBlackTopBlack;
	CUDAArray<AreaStruct> areas = CUDAArray<AreaStruct>(areasSize, 1);
	CUDAArray<Point> initialPoints = CUDAArray<Point>(areasSize, 1);
	AreaStruct initialValue = MakeInitialValue(areas, initialPoints, areasSize);
	hipError_t cudaStatus;

	dim3 blockSize = dim3(defaultThreadCount,defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(maskX,defaultThreadCount), ceilMod(maskY,defaultThreadCount));
	
	for (int i = 0; i < maskX; i++)
    {
		for (int j = 0; j < maskY; j++)
        {
			if (!mask.At(i, j) && isBlack || mask.At(i, j) && !isBlack)
            {
				continue;
            }
			
			isLeftBlackTopImage = IsLeftBlackTopImage(i, j, mask.At(i, j - 1), mask.At(i - 1, j), isBlack);
			isLeftImageTopBlack = IsLeftImageTopBlack(i, j, mask.At(i, j - 1), mask.At(i - 1, j), isBlack);
			isLeftBlackTopBlack = IsLeftBlackTopBlack(i, j, mask.At(i, j - 1), mask.At(i - 1, j), isBlack);

			if (isLeftBlackTopBlack)
            {
				MergeAreas(areas, initialValue, maskX, areasSize, i, j, areaIndex);
				areaIndex--;

				continue;
            }

			if (isLeftBlackTopImage || isLeftImageTopBlack)
            {
				if (isLeftBlackTopImage)
				{
					fillArea<<<gridSize, blockSize>>>(areas, areasSize, maskX, i-1, j, i, j, true);
				}
				else
				{
					fillArea<<<gridSize, blockSize>>>(areas, areasSize, maskX, i, j-1, i, j, false);
				}

                continue;
            }

			Point newPoint = {i, j};
			Point emptyPoint = {0, 0};
			initialPoints.SetAt(0,1, newPoint);
			AreaStruct newArea = {areaIndex, 1, initialPoints.GetData()};
			areas.SetAt(areaIndex++, 1, newArea);
			initialPoints.SetAt(0,1, emptyPoint);
		}
	}

	initialPoints.Dispose();
	
	return areas;
} 

__global__ void changeColor(CUDAArray<bool> mask, CUDAArray<Point> toRestore, int toRestoreCounter)
{
	// coordinates of points in dev_toRestores
	int columnX = blockIdx.x*blockIdx.y*blockDim.x+threadIdx.y*blockDim.x + threadIdx.x;  
	Point point = toRestore.At(columnX, 1);
	
	mask.SetAt(point.X, point.Y, !(mask.At(point.X, point.Y)));
}

CUDAArray<bool> FillAreas(CUDAArray<AreaStruct> areas, CUDAArray<bool> mask, int maskX, int maskY, int threshold)
{
	int maskSize = maskX*maskY + 1;
	int toRestoreCounter = 0;
	int newRestorePoints = 0;
	hipError_t cudaStatus;
	CUDAArray<Point> toRestore = CUDAArray<Point>(maskSize, 1);
		
	for(int i = 0; i < maskSize; i++)
	{
		newRestorePoints = 0;

		if ((areas.At(i,1)).AreaSize < threshold && 
			!IsNearBorder((areas.At(i,1)).Points, (areas.At(i,1)).AreaSize, maskX, maskY))
        {
			while(newRestorePoints <= (areas.At(i,1)).AreaSize)
			{
				toRestore.SetAt(toRestoreCounter, 1, (areas.At(i, 1)).Points[newRestorePoints]); 
				toRestoreCounter++;
			}
		}
	}

	dim3 blockSize = dim3(defaultThreadCount, defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(toRestoreCounter, defaultThreadCount));

	changeColor<<<gridSize, blockSize>>>(mask, toRestore, toRestoreCounter);
	toRestore.Dispose();

	return mask;
}

CUDAArray<bool> PostProcessing(CUDAArray<bool> mask, int maskX, int maskY, int threshold)
{
	CUDAArray<AreaStruct> blackAreas = GenerateAreas(mask, maskX, maskY, true);
	mask = FillAreas(blackAreas, mask, maskX, maskY, threshold);
	CUDAArray<AreaStruct> imageAreas = GenerateAreas(mask, maskX, maskY, false);
	mask = FillAreas(imageAreas, mask, maskX, maskY, threshold);

	return mask;
}
*/
__global__ void cudaGetMagnitude(CUDAArray<float> magnitude, CUDAArray<float> xGradient, CUDAArray<float> yGradient)
{
	int row = defaultRow();
	int column = defaultColumn();
	float newValue = xGradient.At(row,column)*xGradient.At(row,column) +yGradient.At(row,column)*yGradient.At(row,column);
	newValue = sqrt(newValue);
	magnitude.SetAt(row,column, newValue);
}

void GetMagnitude(CUDAArray<float> magnitude, CUDAArray<float> xGradient, CUDAArray<float> yGradient)
{
	dim3 blockSize = dim3(defaultThreadCount,defaultThreadCount);
	dim3 gridSize = 
		dim3(ceilMod(magnitude.Width,defaultThreadCount),
		ceilMod(magnitude.Height,defaultThreadCount));

	cudaGetMagnitude<<<gridSize,blockSize>>>(magnitude, xGradient, yGradient);
	hipError_t error = hipDeviceSynchronize();
}

__global__ void cudaGetMask(CUDAArray<float> initialArray, CUDAArray<bool> mask, int blockSize, float average)
{
	float sum = 0;
	for(int i; i<blockSize; i++)
	{
		for(int j; j<blockSize; j++)
		{
			if(defaultRow()*blockSize+j<initialArray.Height&&
				defaultColumn()*blockSize+i<initialArray.Width)
			{
			sum += initialArray.At(defaultRow()*blockSize+j,defaultColumn()*blockSize+i);
			}
		}
	}
	sum = sum/(blockSize*blockSize);
	mask.SetAt(defaultRow(),defaultColumn(),!(sum < average));
}

float GetAverageFromArray(CUDAArray<float> arrayToAverage)
{
	float sum = 0;
	float* ar = arrayToAverage.GetData();
	for(int i; i<arrayToAverage.Width; i++)
	{		
		for(int j; j<arrayToAverage.Height; j++)
		{
			sum+= ar[i+j*arrayToAverage.Width];
		}
	}
	return sum/(float)(arrayToAverage.Height*arrayToAverage.Width);
	free(ar);
}

CUDAArray<float> loadImage(const char* name, bool sourceIsFloat = false)
{
	FILE* f = fopen(name,"rb");
			
	int width;
	int height;
	
	fread(&width,sizeof(int),1,f);
			
	fread(&height,sizeof(int),1,f);
	
	float* ar2 = (float*)malloc(sizeof(float)*width*height);

	if(!sourceIsFloat)
	{
		int* ar = (int*)malloc(sizeof(int)*width*height);
		fread(ar,sizeof(int),width*height,f);
		for(int i=0;i<width*height;i++)
		{
			ar2[i]=ar[i];
		}
		
		free(ar);
	}
	else
	{
		fread(ar2,sizeof(float),width*height,f);
	}
	
	fclose(f);

	CUDAArray<float> sourceImage = CUDAArray<float>(ar2,width,height);

	free(ar2);		

	return sourceImage;
}

void SaveMask(CUDAArray<bool> mask, const char* name)
{
	FILE* f = fopen(name,"w");
	bool* maskOnCPU = mask.GetData();
	int* ar = (int*)malloc(sizeof(char)*(mask.Width*2+1)*mask.Height);
	int k =0;
	for(int i =0; i<mask.Height; i++)
	{
		for(int j =0; j<mask.Width; j++)
		{
			ar[k++] = (char)maskOnCPU[j+i*mask.Width]?49:48;
			ar[k++] = ' ';
		}
		ar[k++] = '\n';
	}
	fwrite(ar,sizeof(char),(mask.Width*2+1)*mask.Height,f);
	fclose(f);
}

  int main()
  {
	  hipSetDevice(0);

	  //parameters
	  float weightConstant = 0.3; 
	  int windowSize = 12;
	  int threshold = 5;
	  hipError_t cudaStatus;

	  //source image
	  CUDAArray<float> source = loadImage("C:\\temp\\104_6.bin");

	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("CUDAArray<float> source = loadImage(...) - ERROR!!!\n");
	  }

	  int xSizeImg = source.Width;		  
	  int ySizeImg = source.Height;
	
	  source.Dispose();

	  // Sobel:	  
	  CUDAArray<float> xGradient = CUDAArray<float>(xSizeImg,ySizeImg);
	 
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create xGradient[] - ERROR!!!\n");
	  }

	 // SaveArray(cudaImg,"C:\\temp\\check.bin");
	  CUDAArray<float> yGradient = CUDAArray<float>(xSizeImg,ySizeImg);

	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create yGradient[] - ERROR!!!\n");
	  }

	  float xKernelCPU[3][3] = {{-1,0,1},
							{-2,0,2},
							{-1,0,1}};
	  CUDAArray<float> xKernel = CUDAArray<float>(*xKernelCPU,3,3);
	  
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create xKernel[] - ERROR!!!\n");
	  }

	  float yKernelCPU[3][3] = {{-1,-2,-1},
							{0,0,0},
							{1,2,1}};
	  CUDAArray<float> yKernel = CUDAArray<float>(*yKernelCPU,3,3);
	  
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create yKernel[] - ERROR!!!\n");
	  }

	  Convolve(xGradient, source, xKernel);
	  Convolve(yGradient, source, yKernel);

	  //magnitude of gradient
	  CUDAArray<float> magnitude = CUDAArray<float>(xSizeImg,ySizeImg);

	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create magnitude[] - ERROR!!!\n");
	  }

	  xGradient.Dispose();
	  yGradient.Dispose();
	  xKernel.Dispose();
	  yKernel.Dispose();

	  //average magnitude 
	  float average = GetAverageFromArray(magnitude);

	  //dementions of mask
	  int N = (int)ceil(((double)source.Width) / windowSize);
	  int M = (int)ceil(((double)source.Height) / windowSize);
	  
	  //thread configuration in CUDA
	  	dim3 blockSize = dim3(defaultThreadCount,defaultThreadCount);
		dim3 gridSize =dim3(ceilMod(N,defaultThreadCount),
							ceilMod(M,defaultThreadCount));

		//mask creation
		CUDAArray<bool> mask = CUDAArray<bool>(N,M);
		cudaGetMask<<<gridSize, blockSize>>>(magnitude, mask, windowSize, average*weightConstant);
	
		cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("cudaGetMask - ERROR!!!\n");
	  }
	  
	  magnitude.Dispose();

	  //mask = PostProcessing(mask, N, M, threshold);

		//save mask
		SaveMask(mask, "C:\\temp\\mask.txt");
		mask.Dispose();
		hipDeviceReset();

		return 0;
}


