#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "ConvolutionHelper.h"
#include <stdio.h>

#define ceilMod(x, y) (x+y-1)/y
/*
typedef struct 
	{
		int X;
		int Y;
		Point* Next;
	} Point;

typedef struct
	{
		int AreaNumber;
		int AreaSize;
		Point* Points;
		AreaStruct* Next;
	} AreaStruct;
  		
bool IsNearBorder(Point* points, int size, int xBorder, int yBorder)
{
	for (int i = 0; i < size; i++)
    {
		if (points[i].X == 0 || 
			points[i].Y == 0 ||
			points[i].X == xBorder || 
			points[i].Y == yBorder)
		{
			return true;
		}
	}

	return false;
}

void AddPointToArea(AreaStruct* areas, int areaNumber, Point newPoint)
{
	AreaStruct area = areas[areaNumber];

	area.Points[area.AreaSize] = newPoint;
	area.AreaSize++;
	areas[areaNumber] = area;
}

void MergeAreas(CUDAArray<AreaStruct> cudaAreas, int maskX, int areasSize, int i, int j, int areaIndex)
{
	int areaNumberi = 0;
	int areaNumberj = 0;
	AreaStruct* areas = cudaAreas.GetData();
	AreaStruct area = areas[j*maskX + i, 1];

	for (int k = 0; k < areasSize; k++)
	{
		if (area.Points[k].X == i && area.Points[k].Y == j-1)
		{
			areaNumberj = k;
		}
		if (area.Points[k].X == i-1 && area.Points[k].Y == j)
		{
			areaNumberi = k;
		}
	}
                       
	if (areaNumberi != areaNumberj)
	{
		for (int k = 0; k < areas[areaNumberj].AreaSize; k++)
		{
			AddPointToArea(areas, areaNumberi, areas[areaNumberj].Points[k]); 
		}

		for (int k = areaNumberj + 1; k < areaIndex; k++)
		{
			areas[k-1] = areas[k];
		}
	}
    
	Point p = {i,j};
	AddPointToArea(areas, areaNumberi, p); 
	cudaAreas = CUDAArray<AreaStruct>(areas, cudaAreas.Width + 1, cudaAreas.Height);
}

bool IsLeftImageTopBlack(int i, int j, bool topValue, bool leftValue, bool isBlack) 
{
	return (j - 1 >= 0 && (topValue || isBlack) && !(topValue && isBlack) &&					//top block is black 
           (i - 1 >= 0 && (leftValue || !isBlack) && !(leftValue && !isBlack)) || i - 1 < 0);	//left block is not black or not exist
}

bool IsLeftBlackTopImage(int i, int j, bool topValue, bool leftValue, bool isBlack) 
{
	return (i - 1 >= 0 && (leftValue || isBlack) && !(leftValue && isBlack) &&					//left block is black
           (j - 1 >= 0 && (topValue || !isBlack) && !(topValue && !isBlack)) || j - 1 < 0);	    //top block is not black or not exist
}

bool IsLeftBlackTopBlack(int i, int j, bool topValue, bool leftValue, bool isBlack)
{
	return (j - 1 >= 0 && (topValue || isBlack) && !(topValue && isBlack) &&					//top block is black 
            i - 1 >= 0 && (leftValue || isBlack) && !(leftValue && isBlack));					//left block is black
}

void fillArea(AreaStruct* areas, int areasIndex, int maskX, int maskY, int iSearch, int jSearch, int i, int j, bool isFirst)
{
	//int columnX = threadIdx.y*blockDim.x + threadIdx.x; 

	//if (columnX >= areasIndex)
	//{
		//return;
	//}
    
	AreaStruct area = areas[0];
	AreaStruct toSetArea;

	/*hipError_t cudaStatus = hipMalloc(&area, sizeof(AreaStruct));
	if (cudaStatus != hipSuccess) 
	{
		printf("hipMalloc(&area, sizeof(AreaStruct)); - ERROR!!!\n");
	}

	cudaStatus = hipMalloc(&toSetArea, sizeof(AreaStruct));
	if (cudaStatus != hipSuccess) 
	{
		printf("hipMalloc(&toSetArea, sizeof(AreaStruct)); - ERROR!!!\n");
	}*/

/*	cudaStatus = hipMemcpy(area, areas[columnX], sizeof(AreaStruct), hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess) 
	{
		printf("hipMemcpy(area, areas.At(columnX, 1);, sizeof(AreaStruct), hipMemcpyDeviceToDevice); - ERROR!!!\n");
	}

	////////////////////////
	while (area != 0)
	{
		for (int i = 0; i < area.AreaSize; i++)
		{
			if (area.Points[i].X == iSearch && area.Points[i].Y == jSearch)
			{
			 
				area.Points[area.AreaSize + 1].X = i;
				area.Points[area.AreaSize + 1].Y = j;
				area.AreaNumber++;
				toSetArea = areas.At(columnX*maskX + rowY, 1);
				toSetArea.Points = area.Points;
				areas.SetAt(columnX*maskX + rowY, 1, toSetArea);
				return;
			}
		}
	}
}

CUDAArray<AreaStruct> InitializeAreas(CUDAArray<AreaStruct> cudaAreas, int areasSize)
{
	AreaStruct* areas = cudaAreas.GetData();
	AreaStruct area;
	Point* points;

	hipError_t cudaStatus = hipMalloc(&area, sizeof(AreaStruct));
	if (cudaStatus != hipSuccess) 		
	{
		printf("hipMalloc(&area, sizeof(AreaStruct)); - ERROR!!!\n");
	}

	cudaStatus = hipMalloc(&points, sizeof(Point) * areaSize);
	if (cudaStatus != hipSuccess) 		
	{
		printf("hipMalloc(&point, sizeof(Point) * areaSize); - ERROR!!!\n");
	}

	area = {-1, 0, points};

	for (int i = 0; i < areasSize; i++)
	{
		cudaStatus = hipMemcpy(areas[i], area, sizeof(AreaStruct), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) 		
		{
			printf("hipMemcpy(areas[i], area, sizeof(AreaStruct), hipMemcpyHostToDevice); - ERROR!!!\n");
		}
	}
}

AreaStruct* GenerateAreas(bool* mask, int maskX, int maskY, bool isBlack)
{
	//int areasSize = maskX * maskY + 1;
	int areaIndex = 0;
	bool isLeftImageTopBlack = false, isLeftBlackTopImage = false, isLeftBlackTopBlack = false;
	//bool* mask = cudaMask.GetData();
	//CUDAArray<AreaStruct> cudaAreas = CUDAArray<AreaStruct>(areasSize, 1);

	//InitializeAreas(cudaAreas, areasSize);

	for (int i = 0; i < maskX; i++)
    {
		for (int j = 0; j < maskY; j++)
        {
			if (!mask[i, j] && isBlack || mask[i, j] && !isBlack)
            {
				continue;
            }
			
			isLeftBlackTopImage = IsLeftBlackTopImage(i, j, mask[i, j - 1], mask[i - 1, j], isBlack);
			isLeftImageTopBlack = IsLeftImageTopBlack(i, j, mask[i, j - 1], mask[i - 1, j], isBlack);
			isLeftBlackTopBlack = IsLeftBlackTopBlack(i, j, mask[i, j - 1], mask[i - 1, j], isBlack);

			if (isLeftBlackTopBlack)
            {
				MergeAreas(cudaAreas, maskX, areasSize, i, j, areaIndex);
				areaIndex--;
				continue;
            }
			//hipError_t cudaStatus;
			if (isLeftBlackTopImage || isLeftImageTopBlack)
            {
				//dim3 blockSize = dim3(defaultThreadCount,defaultThreadCount);
				//dim3 gridSize = dim3(ceilMod(areaIndex,defaultThreadCount));

				if (isLeftBlackTopImage)
				{

					fillArea(cudaAreas, areaIndex, maskX, maskY, i-1, j, i, j);
				}
				else
				{
					fillArea(cudaAreas, areaIndex, maskX, maskY, i, j-1, i, j);
				}
				//cudaStatus = hipDeviceSynchronize();
				//cudaStatus = hipGetLastError();

                continue;
            }
			

			Point* initialPoints;
			 cudaStatus = hipMalloc(&initialPoints, sizeof(Point) * areasSize);
				cudaStatus = hipDeviceSynchronize();
				cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) 
			{
				printf("hipMalloc(&initialPoints, sizeof(Point) * areasSize); - ERROR!!!\n");
			}

			Point newPoint = {i, j};
			cudaStatus = hipMemcpy(initialPoints, &newPoint, sizeof(Point), hipMemcpyHostToDevice);
			//initialPoints[0] =  newPoint;

			AreaStruct newArea = {areaIndex, 1, initialPoints};
			AreaStruct*	areas = cudaAreas.GetData();
			areas[areaIndex++] = newArea;

			cudaAreas.Dispose();
			cudaAreas = CUDAArray<AreaStruct>(areas, areasSize, 1);
			hipFree(initialPoints);
			hipFree(areas);
		}
	}
	

	return cudaAreas;
} 

//__global__ void changeColor(CUDAArray<bool> mask, CUDAArray<Point> toRestore, int toRestoreCounter)
//{
//	// coordinates of points in dev_toRestores
//	int columnX = blockIdx.x*blockIdx.y*blockDim.x+threadIdx.y*blockDim.x + threadIdx.x; 
//
//	if (columnX >= toRestoreCounter)
//	{
//		return;
//	}
//
//	Point point = toRestore.At(columnX, 1);
//	
//	mask.SetAt(point.X, point.Y, !(mask.At(point.X, point.Y)));
//}

CUDAArray<bool> FillAreas(CUDAArray<AreaStruct> cudaAreas, CUDAArray<bool> cudaMask, int maskX, int maskY, int threshold)
{
	int maskSize = maskX*maskY + 1;
	int toRestoreCounter = 0;
	int newRestorePoints = 0;
	hipError_t cudaStatus;
	AreaStruct* areas = cudaAreas.GetData();
	Point* toRestore;

	cudaStatus = hipMalloc(&toRestore, sizeof(Point) * maskSize);

	if (cudaStatus != hipSuccess) 
	{
		printf("hipMalloc(&toRestore, sizeof(Point) * maskSize); - ERROR!!!\n");
	}
		
	for(int i = 0; i < maskSize; i++)
	{
		newRestorePoints = 0;

		if (areas[i].AreaSize < threshold && 
			!IsNearBorder(areas[i].Points, areas[i].AreaSize, maskX, maskY))
        {
			while(newRestorePoints <= areas[i].AreaSize)
			{
				toRestore[toRestoreCounter] = areas[i].Points[newRestorePoints]; 
				toRestoreCounter++;
			}
		}
	}

	dim3 blockSize = dim3(defaultThreadCount, defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(toRestoreCounter, defaultThreadCount));
	CUDAArray<Point> cudaToRestore = CUDAArray<Point>(toRestore, maskSize, 1);

	changeColor<<<gridSize, blockSize>>>(cudaMask, cudaToRestore, toRestoreCounter);
	cudaToRestore.Dispose();
	hipFree(toRestore);

	return cudaMask;
}

void PostProcessing(bool* mask, int maskX, int maskY, int threshold)
{
	AreaStruct* blackAreas = GenerateAreas(mask, maskX, maskY, true);
	cudaMask = FillAreas(blackAreas, mask, maskX, maskY, threshold);
	AreaStruct* imageAreas = GenerateAreas(mask, maskX, maskY, false);
	cudaMask = FillAreas(imageAreas, mask, maskX, maskY, threshold);
}
*/
__global__ void cudaGetMagnitude(CUDAArray<float> magnitude, CUDAArray<float> xGradient, CUDAArray<float> yGradient)
{
	int row = defaultRow();
	int column = defaultColumn();
	float newValue = xGradient.At(row,column)*xGradient.At(row,column) +yGradient.At(row,column)*yGradient.At(row,column);
	newValue = sqrt(newValue);
	magnitude.SetAt(row,column, newValue);
}

void GetMagnitude(CUDAArray<float> magnitude, CUDAArray<float> xGradient, CUDAArray<float> yGradient)
{
	dim3 blockSize = dim3(defaultThreadCount,defaultThreadCount);
	dim3 gridSize = 
		dim3(ceilMod(magnitude.Width,defaultThreadCount),
		ceilMod(magnitude.Height,defaultThreadCount));

	cudaGetMagnitude<<<gridSize,blockSize>>>(magnitude, xGradient, yGradient);
	hipError_t error = hipDeviceSynchronize();
}

__global__ void cudaGetMask(CUDAArray<float> initialArray, CUDAArray<bool> mask, int blockSize, float average)
{
	if(defaultRow()<mask.Height&&defaultColumn()<mask.Width)
	{
		float sum = 0.0f;

		int rowOffset = defaultRow() * blockSize;
		int columnOffset = defaultColumn() * blockSize;

		for(int i = 0; i < blockSize; i++)
		{
			if(columnOffset + i < initialArray.Width)
			{
				for(int j = 0; j < blockSize; j++)
				{
					if(rowOffset + j < initialArray.Height)
					{
						sum += initialArray.At(rowOffset + j, columnOffset + i);
					}
		
				}
			}
			
		}
		float avg = sum/(blockSize*blockSize);
		bool result = !(avg < average);
		mask.SetAt(defaultRow(),defaultColumn(),result);
	}
}
float GetAverageFromArray(CUDAArray<float> arrayToAverage)
{
	float sum = 0;
	float* ar = arrayToAverage.GetData();
	for(int i; i<arrayToAverage.Width*arrayToAverage.Height; i++)
	{
		sum += ar[i];
	}
	free(ar);
	return sum/(arrayToAverage.Height*arrayToAverage.Width);
	
}

CUDAArray<float> loadImage(const char* name, bool sourceIsFloat = false)
{
	FILE* f = fopen(name,"rb");
			
	int width;
	int height;
	
	fread(&width,sizeof(int),1,f);
			
	fread(&height,sizeof(int),1,f);
	
	float* ar2 = (float*)malloc(sizeof(float)*width*height);

	if(!sourceIsFloat)
	{
		int* ar = (int*)malloc(sizeof(int)*width*height);
		fread(ar,sizeof(int),width*height,f);
		for(int i=0;i<width*height;i++)
		{
			ar2[i]=ar[i];
		}
		
		free(ar);
	}
	else
	{
		fread(ar2,sizeof(float),width*height,f);
	}
	
	fclose(f);

	CUDAArray<float> sourceImage = CUDAArray<float>(ar2,width,height);

	free(ar2);		

	return sourceImage;
}

void SaveMask(bool* mask,int width, int height, const char* name)
{
	FILE* f = fopen(name,"wb");
	
	char* ar = (char*)malloc(sizeof(char)*(width*2+2)*height);
	int k =0;
	for(int i =0; i<height; i++)
	{
		for(int j =0; j<width; j++)
		{

			ar[k++] = mask[j+i*width]?49:48;
			ar[k++] = ' ';
		}
		ar[k++] = 10;
		ar[k++] = 13;
	}
	//fprintf(
	fwrite(ar, sizeof(char), (width*2+2)*height,f);
	fclose(f);
}

  int main() /*char* pathToImg, float weightConstant, int windowSize, int threshold,//parmeters
	  bool* mask,int* maskWidth, int* maskHight) //result*/
  {
	  
	  //parameters
	  float weightConstant = 0.3; 
	  int windowSize = 12;
	  int threshold = 5;

	  int count = 100500;
	  
	  hipError_t cudaStatus = hipGetDeviceCount(&count);

	  cudaStatus = hipSetDevice(0);
	  
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("CUDAArray<float> source = loadImage(...) - ERROR!!!\n");
	  }
	  //source image
	  CUDAArray<float> source = loadImage("C:\\temp\\104_6.bin");
	  //CUDAArray<float> source = loadImage(pathToImg);
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("CUDAArray<float> source = loadImage(...) - ERROR!!!\n");
	  }

	  int xSizeImg = source.Width;		  
	  int ySizeImg = source.Height;

	  // Sobel:	  
	  CUDAArray<float> xGradient = CUDAArray<float>(xSizeImg,ySizeImg);
	  //SaveArray(xGradient,"C:\\temp\\xGradientEmpty.bin");
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create xGradient[] - ERROR!!!\n");
	  }

	  CUDAArray<float> yGradient = CUDAArray<float>(xSizeImg,ySizeImg);

	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create yGradient[] - ERROR!!!\n");
	  }
	 
	  float xKernelCPU[3][3] = {{-1,0,1},
							{-2,0,2},
							{-1,0,1}};
	  CUDAArray<float> xKernel = CUDAArray<float>((float*)&xKernelCPU,3,3);
	  
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create xKernel[] - ERROR!!!\n");
	  }

	  float yKernelCPU[3][3] = {{-1,-2,-1},
							{0,0,0},
							{1,2,1}};
	  CUDAArray<float> yKernel = CUDAArray<float>((float*)&yKernelCPU,3,3);
	  
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create yKernel[] - ERROR!!!\n");
	  }

	  Convolve(xGradient, source, xKernel);
	  Convolve(yGradient, source, yKernel);
	  SaveArray(xGradient,"C:\\temp\\xGradient.bin");
	  SaveArray(yGradient,"C:\\temp\\yGradient.bin");

	  //magnitude of gradient
	  CUDAArray<float> magnitude = CUDAArray<float>(xSizeImg,ySizeImg);
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Create magnitude[] - ERROR!!!\n");
	  }

	  GetMagnitude(magnitude, xGradient, yGradient);
	  cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("cudaGetMask - ERROR!!!\n");
	  }

	  SaveArray(magnitude,"C:\\temp\\magnitude.bin");
	  xGradient.Dispose();
	  yGradient.Dispose();
	  xKernel.Dispose();
	  yKernel.Dispose();

	  //average magnitude 
	  float average = GetAverageFromArray(magnitude);

	  //dementions of mask
	  int N = (int)ceil(((double)source.Width) / windowSize);
	  int M = (int)ceil(((double)source.Height) / windowSize);
	  
	  //thread configuration in CUDA
	  	dim3 blockSize = dim3(defaultThreadCount,defaultThreadCount);
		dim3 gridSize =dim3(ceilMod(N,defaultThreadCount),
							ceilMod(M,defaultThreadCount));

		//mask creation
		CUDAArray<bool> CUDAmask = CUDAArray<bool>(N,M);
		 if (cudaStatus != hipSuccess) 
	  {
		printf("create Mask - ERROR!!!\n");
	  }

		cudaGetMask<<<gridSize,blockSize>>>(magnitude, CUDAmask, windowSize, average*weightConstant);
	
		cudaStatus = hipGetLastError();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("cudaGetMask - ERROR!!!\n");
	  }
	  cudaStatus = hipDeviceSynchronize();
	  if (cudaStatus != hipSuccess) 
	  {
		printf("Synchronize - ERROR!!!\n");
	  }

	  magnitude.Dispose();

	  bool* mask = CUDAmask.GetData();
	 // *maskWidth = (int)(CUDAmask.Width);
	 // *maskHight = (int)CUDAmask.Height;

	 // PostProcessing(mask, N, M, threshold);

		//save mask
	  SaveMask(mask, (int)(CUDAmask.Width), (int)(CUDAmask.Height), "C:\\temp\\mask.txt");
		
	  CUDAmask.Dispose();
	  hipDeviceReset(); 
	  return 0;
}


