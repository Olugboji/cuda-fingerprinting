#include "hip/hip_runtime.h"
﻿//CUDAThinning
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include "CUDAArray.cuh"

//#include<MinutiaMatching.h>

//hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);
hipError_t addWithCuda(int *picture, int width, int height, int *result);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}

CUDAArray<float> loadImage(const char* name, bool sourceIsFloat = false)
{
	FILE* f = fopen(name,"rb");
			
	int width;
	int height;
	
	fread(&width,sizeof(int),1,f);
			
	fread(&height,sizeof(int),1,f);
	
	float* ar2 = (float*)malloc(sizeof(float)*width*height);

	if(!sourceIsFloat)
	{
		int* ar = (int*)malloc(sizeof(int)*width*height);
		fread(ar,sizeof(int),width*height,f);
		for(int i=0;i<width*height;i++)
		{
			ar2[i]=ar[i];
		}
		
		free(ar);
	}
	else
	{
		fread(ar2,sizeof(float),width*height,f);
	}
	
	fclose(f);

	CUDAArray<float> sourceImage = CUDAArray<float>();
	sourceImage.cpuPt = ar2;
	sourceImage.Width = width;
	sourceImage.Height = height;

	//free(ar2);		

	return sourceImage;
	//return ar2;
}

void SaveArray(float* arTest, int width, int height, const char* fname)
{
	FILE* f = fopen(fname,"wb");
	fwrite(&width,sizeof(int),1,f);
	fwrite(&height,sizeof(int),1,f);
	for(int i=0;i<width*height;i++)
	{
		float value = (float)arTest[i];
		int result = fwrite(&value,sizeof(float),1,f);
		result++;
	}
	fclose(f);
	free(arTest);
}

__device__ int B(int *picture, int x, int y, size_t pitch)        //Ìåòîä Â(Ð) âîçâðàùàåò êîëè÷åñòâî ÷åðíûõ ïèêñåëåé â îêðåñòíîñòè òî÷êè Ð
{
	int rowWidthInElements = pitch/sizeof(size_t);
	//return picture[x + (y - 1)*rowWidthInElements] + picture[x + 1 + (y - 1)*rowWidthInElements] + picture[x + 1 + y*rowWidthInElements] + picture[x + 1 + (y + 1)*rowWidthInElements] +
 //          picture[x * (y + 1)*rowWidthInElements] + picture[x - 1 + (y + 1)*rowWidthInElements] + picture[x - 1 + y*rowWidthInElements] + picture[x - 1 * (y - 1)*rowWidthInElements];

	return picture[x + (y - 1)*rowWidthInElements] + picture[x + 1 + (y - 1)*rowWidthInElements] + picture[x + 1 + y*rowWidthInElements] + picture[x + 1 + (y + 1)*rowWidthInElements] +
		   picture[x + (y + 1)*rowWidthInElements] + picture[x - 1 + (y + 1)*rowWidthInElements] + picture[x - 1 + y*rowWidthInElements] + picture[x - 1 + (y - 1)*rowWidthInElements];
			
}

__device__ int A(int *picture, int x, int y, size_t pitch)        //Ìåòîä À(Ð) âîçâðàùàåò êîëè÷åñòâî ïîäðÿä èäóùèõ áåëûõ è ÷åðíûõ ïèêñåëåé âîêðóã òî÷êè Ð (..0->1..)
{
	int rowWidthInElements = pitch/sizeof(size_t);
	int counter = 0;
    if((picture[x + (y - 1)*rowWidthInElements] == 0) && (picture[x + 1 + (y - 1)*rowWidthInElements] == 1))
    {
        counter++;
    }
    if ((picture[x + 1 + (y - 1)*rowWidthInElements] == 0) && (picture[x + 1 + y*rowWidthInElements] == 1))
    {
        counter++;
    }
    if ((picture[x + 1 + y*rowWidthInElements] == 0) && (picture[x + 1 + (y + 1)*rowWidthInElements] == 1))
    {
        counter++;
    }
    if ((picture[x + 1 + (y + 1)*rowWidthInElements] == 0) && (picture[x + (y + 1)*rowWidthInElements] == 1))
    {
        counter++;
    }
    if ((picture[x + (y + 1)*rowWidthInElements] == 0) && (picture[x - 1 + (y + 1)*rowWidthInElements] == 1))
    {
        counter++;
    }
    if ((picture[x - 1 + (y + 1)*rowWidthInElements] == 0) && (picture[x - 1 + y*rowWidthInElements] == 1))
    {
        counter++;
    }
    if ((picture[x - 1 + y*rowWidthInElements] == 0) && (picture[x - 1 + (y - 1)*rowWidthInElements] == 1))
    {
        counter++;
    }
    if ((picture[x - 1 + (y - 1)*rowWidthInElements] == 0) && (picture[x + (y - 1)*rowWidthInElements] == 1))
    {
        counter++;
    }
    return counter;
}

__global__ void compare(int* pictureToRemove, int* picture, size_t pitch, int width, int height)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
	int rowWidthInElements = pitch/sizeof(size_t);
	
	if((pictureToRemove[y*rowWidthInElements + x] == 0) && (x > 0) && (y > 0) && (x < (width - 1)) && (y < (height - 1)))
	{
		picture[y*rowWidthInElements + x] = 0;
		pictureToRemove[y*rowWidthInElements + x] = 1;
	}

}

__global__ void ThiningImgWithCUDA(CUDAArray<int> thinnedPicture, int width, int height)
{
	int column = defaultColumn();
	int row = defaultRow();
	thinnedPicture.SetAt(row, column, 1);
	//if((x > 0) && (y > 0) && (x < (width - 1)) && (y < (height - 1)))
	//{
	//	if ((picture[j, i] == 1) && (2 <= B(picture, j, i)) && (B(picture, j, i) <= 6) && (A(picture, j, i) == 1) &&     //Непосредственное удаление точки, см. Zhang-Suen thinning algorithm, http://www-prima.inrialpes.fr/perso/Tran/Draft/gateway.cfm.pdf
 //                       (picture[j, i - 1]*picture[j + 1, i]*picture[j, i + 1] == 0) &&
 //                       (picture[j + 1, i]*picture[j, i + 1]*picture[j - 1, i] == 0))
 //                   {
 //                       picture[j, i] = 0;
 //                   }
	//}

}


__global__ void ThiningPictureWithCUDA(int* newPicture, int *picture ,size_t pitch, int width, int height,bool* hasChanged)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
	int rowWidthInElements = pitch/sizeof(size_t);
    //if((x > 0) && (y > 0) && (x < (width - 1)) && (y < (height - 1)))
	if((x > 0) && (y > 0) && (x < (width - 1)) && (y < (height - 1)))
	{             
		if ((picture[x + y*rowWidthInElements] == 1) && (2 <= B(picture, x, y, pitch)) && (B(picture, x, y, pitch) <= 6) && (A(picture, x, y, pitch) == 1) &&                         
			 (picture[x + (y - 1)*rowWidthInElements] * picture[x + 1 + y*rowWidthInElements] * picture[x + (y + 1)*rowWidthInElements] == 0) &&
             (picture[x + 1 + y*rowWidthInElements] * picture[x + (y + 1)*rowWidthInElements] * picture[x - 1 + y*rowWidthInElements] == 0))
         {
				newPicture[x + y*rowWidthInElements] = 0;
                hasChanged[0] = true;
		 }	
		
	}
	//newPicture[x+rowWidthInElements*y] = picture[x+rowWidthInElements*y];
}

__global__ void ThiningPictureWithCUDA2(int* newPicture, int *picture ,size_t pitch, int width, int height, bool* hasChanged)
{
	//int *picture = newPicture;
	int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
	int rowWidthInElements = pitch/sizeof(size_t);
	if((x > 0) && (y > 0) && (x < (width - 1)) && (y < (height - 1)))
	{             
		if ((picture[x + y*rowWidthInElements] == 1) && (2 <= B(picture, x, y, pitch)) && (B(picture, x, y, pitch) <= 6) && (A(picture, x, y, pitch) == 1) &&
			(picture[x + (y - 1)*rowWidthInElements] * picture[x + 1 + y*rowWidthInElements] * picture[x - 1 + y*rowWidthInElements] == 0) &&
			(picture[x + (y - 1)*rowWidthInElements] * picture[x + (y + 1)*rowWidthInElements] * picture[x - 1 + y*rowWidthInElements] == 0))
		{
			newPicture[x + y*rowWidthInElements] = 0;
			hasChanged[0] = true;
		} 
	}
	//newPicture[x+rowWidthInElements*y] = picture[x+rowWidthInElements*y];

}

//Doesn't work correctly with parallel method
//__global__ void ThiningPictureWithCUDA3(int* newPicture, int *picture ,size_t pitch, int width, int height)
//{
//	int x = threadIdx.x + blockIdx.x*blockDim.x;
//    int y = threadIdx.y + blockIdx.y*blockDim.y;
//	int rowWidthInElements = pitch/sizeof(size_t);
//	if((x > 0) && (y > 0) && (x < (width - 1)) && (y < (height - 1)))
//	{           
//		if ((picture[x + y*rowWidthInElements] == 1) &&
//		   (((picture[x + (y - 1)*rowWidthInElements] * picture[x + 1 + y*rowWidthInElements] == 1) && (picture[x - 1 + (y + 1)*rowWidthInElements] != 1)) || ((picture[x + 1 + y*rowWidthInElements] * picture[x + (y + 1)*rowWidthInElements] == 1) && (picture[x - 1 + (y - 1)*rowWidthInElements] != 1)) ||      //Небольшая модификцаия алгоритма для ещё большего утоньшения
//           (( picture[x + (y + 1)*rowWidthInElements] * picture[x - 1 + y*rowWidthInElements] == 1) && (picture[x + 1 + (y - 1)*rowWidthInElements] != 1)) || ((picture[x + (y - 1)*rowWidthInElements] * picture[x - 1 + y*rowWidthInElements] == 1) && (picture[x + 1 + (y + 1)*rowWidthInElements] != 1))))
//        {
//			newPicture[x + y*rowWidthInElements] = 0;
//        }
//	}else
//	{
//		newPicture[x+rowWidthInElements*y] = picture[x+rowWidthInElements*y];
//	}
//}

void DeleteCorners(int *picture, int width, int height)
{
	
	//int x = threadIdx.x + blockIdx.x*blockDim.x;
    //int y = threadIdx.y + blockIdx.y*blockDim.y;
	int rowWidthInElements = width;
	for(int x = 1; x < width - 1; x++)
	{
		for(int y = 1; y < height - 1; y++)
		{
			if ((picture[x + y*rowWidthInElements] == 1) &&
				(((picture[x + (y - 1)*rowWidthInElements] * picture[x + 1 + y*rowWidthInElements] == 1) && (picture[x - 1 + (y + 1)*rowWidthInElements] != 1)) || ((picture[x + 1 + y*rowWidthInElements] * picture[x + (y + 1)*rowWidthInElements] == 1) && (picture[x - 1 + (y - 1)*rowWidthInElements] != 1)) ||      //Небольшая модификцаия алгоритма для ещё большего утоньшения
				(( picture[x + (y + 1)*rowWidthInElements] * picture[x - 1 + y*rowWidthInElements] == 1) && (picture[x + 1 + (y - 1)*rowWidthInElements] != 1)) || ((picture[x + (y - 1)*rowWidthInElements] * picture[x - 1 + y*rowWidthInElements] == 1) && (picture[x + 1 + (y + 1)*rowWidthInElements] != 1))))
			{
				picture[x + y*rowWidthInElements] = 0;
			}
		}
	}
}








int main()
{
	//int size = 32;
	int width; //= size;
	int	height; //= size;
	CUDAArray<float> img = loadImage("C:\\temp\\104_6_Binarizated.bin", true);
	width = img.Width;
	height = img.Height;
	int *picture = (int*)malloc(width*height*sizeof(int));
	int *result = (int*)malloc(width*height*sizeof(int));
	float* picture1;// = (float*)malloc(width*height*sizeof(float));
	FILE *in = fopen("C:\\Users\\CUDA Fingerprinting2\\picture.in","r");
	FILE *out = fopen("C:\\Users\\CUDA Fingerprinting2\\picture.out","w");
	float* result1 = (float*)malloc(width*height*sizeof(float));

	picture1 = img.cpuPt;
	for(int i = 0; i < width; i++)
	{
		for(int j = 0; j < height; j++)
		{
			picture[j*width + i] = picture1[j*width + i] > 0 ? 0 : 1;
		}
	}

	//for(int i = 0; i < width; i++)
	//{
	//	for(int j = 0; j < height; j++)
	//	{
	//		printf("%.0f ",picture1[j*width + i]);
	//	}
	//	printf("\n");
	//}
	//printf("\n");

	//for(int i = 0; i < width; i++)
	//{
	//	for(int j = 0; j < height; j++)
	//	{
	//		fscanf(in,"%d",&picture[j*size + i]);
	//	}
	//}

	//for(int i = 0; i < width; i++)
	//{
	//	for(int j = 0; j < height; j++)
	//	{
	//		printf("%d ",picture[j*size + i]);
	//	}
	//	printf("\n");
	//}
	//printf("\n");

    hipError_t cudaStatus = addWithCuda(picture, width, height, result); 
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	for(int i = 0; i < width; i++)
	{
		for(int j = 0; j < height; j++)
		{
			fprintf(out,"%d ",result[j*width + i]);
		}
		fprintf(out,"\n");
	}
	for(int i = 0; i < width; i++)
	{
		for(int j = 0; j < height; j++)
		{
			result1[j*width + i] = result[j*width + i] > 0 ? 0 : 255;
		}
	}
	SaveArray(result1, width, height,"C:\\temp\\104_6_BinarizatedThinnedCUDA.bin");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }

	free(picture);
	free(result);
	img.Dispose();
	free(picture1);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *picture, int width, int height, int *result)
{
	int* dev_picture; 
	int* dev_pictureThinned;
	int* dev_pictureToRemove;
	bool hasChanged;
	bool* dev_hasChanged;
	int *pictureToRemove = (int*)malloc(width*height*sizeof(int));
	for(int i = 0; i < width; i++)
	{
		for(int j = 0; j < height; j++)
		{
			pictureToRemove[j*width + i] = 1;
		}
	}
	//CUDAArray<int> img = CUDAArray<int>(picture, width, height);
	//CUDAArray<int> imgout = CUDAArray<int>(dev_pictureThinned, width, height);
//	img.
    hipError_t cudaStatus;
	size_t pitch;
    size_t pitch1;
	size_t pitch2;
	
	cudaStatus = hipMallocPitch((void**)&dev_pictureToRemove, &pitch2, width*sizeof(int), height);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocPitch");
        goto Error;
    }
	cudaStatus = hipMemcpy2D(dev_pictureToRemove, pitch2, pictureToRemove, width*sizeof(int), width*sizeof(int), height, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy!");
        goto Error;
    }
	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	//Allocate GPU buffers for picture.
	cudaStatus = hipMallocPitch((void**)&dev_picture, &pitch, width*sizeof(int), height);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocPitch!");
        goto Error;
    }
	cudaStatus = hipMalloc((void**)&dev_hasChanged, sizeof(bool));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc!");
        goto Error;
    }

	cudaStatus = hipMallocPitch((void**)&dev_pictureThinned, &pitch1, width*sizeof(int), height);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocPitch!");
        goto Error;
    }

    // Copy input picture from host memory to GPU buffers.

    cudaStatus = hipMemcpy2D(dev_picture, pitch, picture, width*sizeof(int), width*sizeof(int), height, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    int dimA = width*height;
    int numThreadsPerBlock = 16;
    int numBlocks = dimA / numThreadsPerBlock;
    
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);

	//for(int i = 0; i < pitch1/sizeof(size_t); i++)
	//{
	//	for(int j = 0; j < height; j++)
	//	{	

	//			dev_pictureThinned[i*(pitch1/sizeof(size_t)) + j] = 1;

	//	}
	//}

	do{
		hasChanged = false;
		cudaStatus = hipMemcpy(dev_hasChanged, &hasChanged, sizeof(bool), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		ThiningPictureWithCUDA<<<dim3(ceilMod(width,16),ceilMod(height,16)),dim3(16,16)>>>(dev_pictureToRemove, dev_picture, pitch, width, height, dev_hasChanged);

		compare<<<dim3(ceilMod(width,16),ceilMod(height,16)),dim3(16,16)>>>(dev_pictureToRemove, dev_picture, pitch, width, height);

		
		cudaStatus = hipMemcpy2D(result, width*sizeof(int), dev_picture, pitch, width*sizeof(int), height, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		//for(int i = 0; i < width; i++)
		//{
		//	for(int j = 0; j < height; j++)
		//	{
		//		printf("%d ",result[j*width + i]);
		//	}
		//	printf("\n");
		//}
		//printf("\n");
		
		ThiningPictureWithCUDA2<<<dim3(ceilMod(width,16),ceilMod(height,16)),dim3(16,16)>>>(dev_pictureToRemove, dev_picture, pitch, width, height, dev_hasChanged);

		compare<<<dim3(ceilMod(width,16),ceilMod(height,16)),dim3(16,16)>>>(dev_pictureToRemove, dev_picture, pitch, width, height);


		cudaStatus = hipMemcpy2D(result, width*sizeof(int), dev_picture, pitch, width*sizeof(int), height, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		//for(int i = 0; i < width; i++)
		//{
		//	for(int j = 0; j < height; j++)
		//	{
		//		printf("%d ",result[j*width + i]);
		//	}
		//	printf("\n");
		//}
		//printf("\n");

		cudaStatus = hipMemcpy(&hasChanged, dev_hasChanged, sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

	}while(hasChanged);
	

	
	/*ThiningPictureWithCUDA2<<<dim3(2,2),dim3(16,16)>>>(dev_picture, dev_pictureThinned, pitch1, width, height);
	ThiningPictureWithCUDA<<<dim3(2,2),dim3(16,16)>>>(dev_picture, dev_pictureThinned, pitch1, width, height);
	ThiningPictureWithCUDA3<<<dim3(2,2),dim3(16,16)>>>(dev_pictureThinned, dev_picture, pitch, width, height);*/
	
	//ThiningImgWithCUDA<<<dim3(2,2),dim3(16,16)>>>(img, width, height);
	//result = img.GetData();

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
	
	cudaStatus = hipMemcpy2D(result, width*sizeof(int), dev_picture, pitch, width*sizeof(int), height, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	
	DeleteCorners(result, width, height);

    // Copy output vector from GPU buffer to host memory.

	//for(int i = 0; i < width; i++)
	//{
	//	for(int j = 0; j < height; j++)
	//	{
	//		printf("%d ",result[j*width + i]);
	//	}
	//	printf("\n");
	//}

Error:
    hipFree(dev_picture);
    hipFree(dev_pictureThinned);
	hipFree(dev_pictureToRemove);
	hipFree(dev_hasChanged);
	free(pictureToRemove);
	//img.Dispose();
    //hipFree(dev_b);
    
    return cudaStatus;
}
